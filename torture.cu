#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <vector>

/*******/
/* GPU */
/*******/

#define BLOCK_SIZE 256
#define HALF_GB (size_t)(512 * 1024 * 1024)

#define CUDA_CHECK(call) \
    do { \
        hipError_t _cu_error = call; \
        if (_cu_error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(_cu_error)); \
            return _cu_error; \
        } \
    } while(0)

__global__ void gpu_torture_kernel(float **data_blocks, size_t n_blocks, size_t block_size) {
    size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total_elements = n_blocks * block_size;
    
    if (thread_id < total_elements) {
        size_t block_idx = thread_id / block_size;
        size_t element_idx = thread_id % block_size;
        
        float x = 1.0f;
        float y = 2.0f;
        float z = 3.0f;
        
        for (int i = 0; i < 1000000; i++) {
            x = sinf(x) * cosf(y) * tanf(z);
            y = expf(x) * logf(fabsf(y)) * sqrtf(fabsf(z));
            z = powf(x, y) * fmodf(z, 3.14159f);
            
            // Access and modify data from different blocks
            for (size_t j = 0; j < n_blocks; j++) {
                size_t idx = (block_idx + j) % n_blocks;
                float* block = data_blocks[idx];
                float val = block[(element_idx + j) % block_size];
                x += val;
                y *= val;
                z -= val;
            }
        }
        
        // Write results back to all blocks
        for (size_t j = 0; j < n_blocks; j++) {
            size_t idx = (block_idx + j) % n_blocks;
            float* block = data_blocks[idx];
            block[(element_idx + j) % block_size] = x + y + z;
        }
    }
}

hipError_t launch_gpu_torture() {
    std::vector<float*> gpu_memory_blocks;
    size_t total_allocated = 0;
    
    while (true) {
        float* d_data;
        hipError_t err = hipMalloc(&d_data, HALF_GB);
        if (err != hipSuccess) {
            if (err == hipErrorOutOfMemory) {
                hipGetLastError(); // Clear the error
                break;  // We've used all available memory
            }
            return err;
        }
        
        gpu_memory_blocks.push_back(d_data);
        total_allocated += HALF_GB;
    }
    
    printf("Total GPU memory allocated: %.2f GB\n", total_allocated / (1024.0 * 1024.0 * 1024.0));
    
    size_t n_blocks = gpu_memory_blocks.size();
    size_t block_size = HALF_GB / sizeof(float);
    size_t total_elements = n_blocks * block_size;
    
    // Allocate and copy device pointers to GPU
    float **d_data_blocks;
    CUDA_CHECK(hipMalloc(&d_data_blocks, n_blocks * sizeof(float*)));
    CUDA_CHECK(hipMemcpy(d_data_blocks, gpu_memory_blocks.data(), n_blocks * sizeof(float*), hipMemcpyHostToDevice));
    
    dim3 block(BLOCK_SIZE);
    dim3 grid((total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    while (1) {
        gpu_torture_kernel<<<grid, block>>>(d_data_blocks, n_blocks, block_size);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        puts("GPU torture kernel finished");
    }
    
    return hipSuccess;
}

/*******/
/* CPU */
/*******/

void *infinite_loop(void *unused) {
  while (1)
    ;
}

char *store_mem = NULL;

void alloc_mem(int n_gb) {
  if (n_gb <= 0)
    return;

  size_t n_b = (size_t)n_gb * 1000 * 1000 * 1000;
  store_mem = (char *)calloc(1, n_b);
  if (!store_mem)
    puts("malloc() failed."), exit(1);

  for (size_t i = 0; i < n_b; i++)
    store_mem[i] = 0;
}


void torture_cpu(void) {
    int numThreads = sysconf(_SC_NPROCESSORS_ONLN) - 1; // Get the number of processors.
    pthread_t threads[numThreads];
    int rc;
    for (size_t t = 0; t < numThreads; t++) {
        rc = pthread_create(&threads[t], NULL, infinite_loop, NULL);
        if (rc) {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(1);
        }
    }

    // Launch GPU torture in the main thread
    launch_gpu_torture();
}

int main(int argc, char **argv) {
    int n_gb = -1;
    if (argc < 2)
        n_gb = 0;
    else if (argc == 2)
        n_gb = atoi(argv[1]);
    else
        return puts("Usage: load_all_threads <n_gb>"), 1;

    // Initialize CUDA
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        exit(1);
    }
    CUDA_CHECK(hipSetDevice(0));

    alloc_mem(n_gb);
    
    hipError_t result = launch_gpu_torture();
    if (result != hipSuccess) {
        fprintf(stderr, "GPU torture failed: %s\n", hipGetErrorString(result));
        return 1;
    }
    
    torture_cpu();
    return 0;
}
