#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string.h>
#include <stdint.h>
#include <time.h>
/*******/
/* GPU */
/*******/

#define BLOCK_SIZE 256
#define ONE_GB (size_t)(1024 * 1024 * 1024)
#define HALF_GB (ONE_GB / 2)

#define CUDA_CHECK(call) \
    do { \
        hipError_t _cu_error = call; \
        if (_cu_error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(_cu_error)); \
            return _cu_error; \
        } \
    } while(0)

__global__ void gpu_torture_kernel(float **data_blocks, size_t n_blocks, size_t block_size) {
    size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total_elements = n_blocks * block_size;
    
    if (thread_id < total_elements) {
        size_t block_idx = thread_id / block_size;
        size_t element_idx = thread_id % block_size;
        
        float x = 1.0f;
        float y = 2.0f;
        float z = 3.0f;
        
        for (int i = 0; i < 1000000; i++) {
            x = sinf(x) * cosf(y) * tanf(z);
            y = expf(x) * logf(fabsf(y)) * sqrtf(fabsf(z));
            z = powf(x, y) * fmodf(z, 3.14159f);
            
            // Access and modify data from different blocks
            for (size_t j = 0; j < n_blocks; j++) {
                size_t idx = (block_idx + j) % n_blocks;
                float* block = data_blocks[idx];
                float val = block[(element_idx + j) % block_size];
                x += val;
                y *= val;
                z -= val;
            }
        }
        
        // Write results back to all blocks
        for (size_t j = 0; j < n_blocks; j++) {
            size_t idx = (block_idx + j) % n_blocks;
            float* block = data_blocks[idx];
            block[(element_idx + j) % block_size] = x + y + z;
        }
    }
}

void* launch_gpu_torture(void* arg) {
  (void)arg;
    std::vector<float*> gpu_memory_blocks;
    size_t total_allocated = 0;
    
    while (true) {
        float* d_data;
        hipError_t err = hipMalloc(&d_data, HALF_GB);
        if (err != hipSuccess) {
            if (err == hipErrorOutOfMemory) {
                hipGetLastError(); // Clear the error
                break;  // We've used all available memory
            }
            fprintf(stderr, "CUDA error allocating memory: %s\n", hipGetErrorString(err));
            exit(1);
        }
        
        gpu_memory_blocks.push_back(d_data);
        total_allocated += HALF_GB;
    }
    
    printf("Total GPU memory allocated: %.2f GB\n", total_allocated / ONE_GB);
    
    size_t n_blocks = gpu_memory_blocks.size();
    size_t block_size = HALF_GB / sizeof(float);
    size_t total_elements = n_blocks * block_size;
    
    // Allocate and copy device pointers to GPU
    float **d_data_blocks;
    if (hipMalloc(&d_data_blocks, n_blocks * sizeof(float*)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for data blocks\n");
        exit(1);
    }
    if (hipMemcpy(d_data_blocks, gpu_memory_blocks.data(), n_blocks * sizeof(float*), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy data blocks to device\n");
        exit(1);
    }
    
    while (true) {

        // Launch the kernel
        dim3 block(BLOCK_SIZE);
        dim3 grid((total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);
        gpu_torture_kernel<<<grid, block>>>(d_data_blocks, n_blocks, block_size);

        // Check for errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error launching kernel: %s\n", hipGetErrorString(err));
            exit(1);
        } 

        // Wait for the kernel to complete
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error synchronizing devices: %s\n", hipGetErrorString(err));
        } else {
            puts("GPU torture kernel finished");
        }
        
    }
}

/*******/
/* CPU */
/*******/

// Xorshift64 PRNG implementation
struct xorshift64_state {
    uint64_t a;
};

uint64_t xorshift64(struct xorshift64_state *state) {
    uint64_t x = state->a;
    x ^= x << 13;
    x ^= x >> 7;
    x ^= x << 17;
    return state->a = x;
}


char *alloc_mem(size_t n_bytes) {
    if (n_bytes == 0)
        return NULL;

    char *mem = (char *)calloc(1, n_bytes);
    if (!mem) {
        puts("malloc() failed."), exit(1);
    }

    for (size_t i = 0; i < n_bytes; i++)
        mem[i] = 0;

    printf("Allocated %.2f GB of memory\n", n_bytes / (double)ONE_GB);
    return mem;
}

typedef struct {
    char *mem;
    size_t size;
    uint64_t seed;
} ThreadArg;

void *cpu_task(void *arg) {
    ThreadArg *thread_arg = (ThreadArg *)arg;
    char *mem = thread_arg->mem;
    size_t size = thread_arg->size;
    struct xorshift64_state state = {thread_arg->seed};
    
    while (1) {
        for (size_t i = 0; i < size; i++) {
            size_t pos = xorshift64(&state) % size;
            char value = (char)(xorshift64(&state) & 0xFF);
            mem[pos] = value;
            // Force memory access
            volatile char dummy = mem[pos];
            (void)dummy;
        }
    }
    return NULL;
}

void torture_cpu(char *mem, size_t mem_size) {
    int numThreads = sysconf(_SC_NPROCESSORS_ONLN);
    pthread_t threads[numThreads];
    ThreadArg thread_args[numThreads];
    size_t chunk_size = mem_size / numThreads;

    // Use current time as a seed for the first thread
    uint64_t seed = (uint64_t)time(NULL);
    for (int t = 0; t < numThreads; t++) {
        thread_args[t].mem = mem + t * chunk_size;
        thread_args[t].size = (t == numThreads - 1) ? (mem_size - t * chunk_size) : chunk_size;
        thread_args[t].seed = seed + t; // Use a different seed for each thread
        int rc = pthread_create(&threads[t], NULL, cpu_task, &thread_args[t]);
        if (rc) {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(1);
        }
    }

    for (int t = 0; t < numThreads; t++) {
        pthread_join(threads[t], NULL);
    }
}


int main(int argc, char **argv) {
    size_t mem_bytes = 0;
    bool run_cpu = false;
    bool run_gpu = false;

    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--cpu") == 0) {
            run_cpu = true;
        } else if (strcmp(argv[i], "--gpu") == 0) {
            run_gpu = true;
        } else if (strncmp(argv[i], "--mem=", strlen("--mem=")) == 0) {
            char* mem_str = argv[i] + strlen("--mem=");
            char* endptr;
            double mem_value = strtod(mem_str, &endptr);
            if (mem_value <= 0 || mem_str == endptr) {
                fprintf(stderr, "Invalid memory value: %s\n", mem_str);
                return 1;
            }
            if (*endptr == '\0' || strcasecmp(endptr, "B") == 0) {
                mem_bytes = (size_t)mem_value;
            } else if (strcasecmp(endptr, "K") == 0 || strcasecmp(endptr, "KB") == 0) {
                mem_bytes = (size_t)(mem_value * 1024);
            } else if (strcasecmp(endptr, "M") == 0 || strcasecmp(endptr, "MB") == 0) {
                mem_bytes = (size_t)(mem_value * 1024 * 1024);
            } else if (strcasecmp(endptr, "G") == 0 || strcasecmp(endptr, "GB") == 0) {
                mem_bytes = (size_t)(mem_value * 1024 * 1024 * 1024);
            } else {
                fprintf(stderr, "Invalid memory unit: %s\n", endptr);
                return 1;
            }
        } else {
            fprintf(stderr, "Usage: %s [--cpu] [--gpu] [--mem=<size>[B|K|M|G]]\n", argv[0]);
            return 1;
        }
    }

    if (!run_cpu && !run_gpu) {
        fprintf(stderr, "Error: At least one of --cpu or --gpu must be specified.\n");
        return 1;
    }

    char *allocated_mem = alloc_mem(mem_bytes);

    pthread_t gpu_thread;
    if (run_gpu) {
        // Initialize CUDA
        int deviceCount;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error getting device count: %s\n", hipGetErrorString(err));
            return 1;
        }
        if (deviceCount == 0) {
            fprintf(stderr, "No CUDA devices found\n");
            return 1;
        }
        err = hipSetDevice(0);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error setting device: %s\n", hipGetErrorString(err));
            return 1;
        }

        // Create GPU torture thread
        if (pthread_create(&gpu_thread, NULL, launch_gpu_torture, NULL) != 0) {
            fprintf(stderr, "Failed to create GPU torture thread\n");
            return 1;
        }
    }

    if (run_cpu) {
        torture_cpu(allocated_mem, mem_bytes);
    }

    // Wait for GPU thread to finish if it was started
    if (run_gpu) {
        pthread_join(gpu_thread, NULL);
    }

    // Free allocated memory
    free(allocated_mem);

    return 0;
}
