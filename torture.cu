#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string.h>
#include <atomic>

/*******/
/* GPU */
/*******/

#define BLOCK_SIZE 256
#define HALF_GB (size_t)(512 * 1024 * 1024)

#define CUDA_CHECK(call) \
    do { \
        hipError_t _cu_error = call; \
        if (_cu_error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(_cu_error)); \
            return _cu_error; \
        } \
    } while(0)

__global__ void gpu_torture_kernel(float **data_blocks, size_t n_blocks, size_t block_size) {
    size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total_elements = n_blocks * block_size;
    
    if (thread_id < total_elements) {
        size_t block_idx = thread_id / block_size;
        size_t element_idx = thread_id % block_size;
        
        float x = 1.0f;
        float y = 2.0f;
        float z = 3.0f;
        
        for (int i = 0; i < 1000000; i++) {
            x = sinf(x) * cosf(y) * tanf(z);
            y = expf(x) * logf(fabsf(y)) * sqrtf(fabsf(z));
            z = powf(x, y) * fmodf(z, 3.14159f);
            
            // Access and modify data from different blocks
            for (size_t j = 0; j < n_blocks; j++) {
                size_t idx = (block_idx + j) % n_blocks;
                float* block = data_blocks[idx];
                float val = block[(element_idx + j) % block_size];
                x += val;
                y *= val;
                z -= val;
            }
        }
        
        // Write results back to all blocks
        for (size_t j = 0; j < n_blocks; j++) {
            size_t idx = (block_idx + j) % n_blocks;
            float* block = data_blocks[idx];
            block[(element_idx + j) % block_size] = x + y + z;
        }
    }
}

std::atomic<bool> gpu_torture_running(false);

void launch_gpu_torture() {
    gpu_torture_running.store(true);
    std::vector<float*> gpu_memory_blocks;
    size_t total_allocated = 0;
    
    while (true) {
        float* d_data;
        hipError_t err = hipMalloc(&d_data, HALF_GB);
        if (err != hipSuccess) {
            if (err == hipErrorOutOfMemory) {
                hipGetLastError(); // Clear the error
                break;  // We've used all available memory
            }
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            return;
        }
        
        gpu_memory_blocks.push_back(d_data);
        total_allocated += HALF_GB;
    }
    
    printf("Total GPU memory allocated: %.2f GB\n", total_allocated / (1024.0 * 1024.0 * 1024.0));
    
    size_t n_blocks = gpu_memory_blocks.size();
    size_t block_size = HALF_GB / sizeof(float);
    size_t total_elements = n_blocks * block_size;
    
    // Allocate and copy device pointers to GPU
    float **d_data_blocks;
    if (hipMalloc(&d_data_blocks, n_blocks * sizeof(float*)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for data blocks\n");
        return;
    }
    if (hipMemcpy(d_data_blocks, gpu_memory_blocks.data(), n_blocks * sizeof(float*), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy data blocks to device\n");
        return;
    }
    
    dim3 block(BLOCK_SIZE);
    dim3 grid((total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    while (gpu_torture_running.load()) {
        gpu_torture_kernel<<<grid, block>>>(d_data_blocks, n_blocks, block_size);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            break;
        }
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            break;
        }
        puts("GPU torture kernel finished");
    }
    
    // Free allocated memory
    for (auto block : gpu_memory_blocks) {
        hipFree(block);
    }
    hipFree(d_data_blocks);
}

void* gpu_torture_thread(void* arg) {
    launch_gpu_torture();
    return NULL;
}

/*******/
/* CPU */
/*******/

void *infinite_loop(void *unused) {
  while (1)
    ;
}

char *store_mem = NULL;

void alloc_mem(int n_gb) {
  if (n_gb <= 0)
    return;

  size_t n_b = (size_t)n_gb * 1000 * 1000 * 1000;
  store_mem = (char *)calloc(1, n_b);
  if (!store_mem)
    puts("malloc() failed."), exit(1);

  for (size_t i = 0; i < n_b; i++)
    store_mem[i] = 0;
}


void torture_cpu(void) {
    int numThreads = sysconf(_SC_NPROCESSORS_ONLN) - 1; // Get the number of processors.
    pthread_t threads[numThreads];
    int rc;
    for (size_t t = 0; t < numThreads; t++) {
        rc = pthread_create(&threads[t], NULL, infinite_loop, NULL);
        if (rc) {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(1);
        }
    }

    // Launch GPU torture in the main thread
    launch_gpu_torture();
}

int main(int argc, char **argv) {
    int n_gb = 0;
    bool run_cpu = false;
    bool run_gpu = false;

    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--cpu") == 0) {
            run_cpu = true;
        } else if (strcmp(argv[i], "--gpu") == 0) {
            run_gpu = true;
        } else if (strncmp(argv[i], "--mem=", 6) == 0) {
            n_gb = atoi(argv[i] + 6);
        } else {
            fprintf(stderr, "Usage: %s [--cpu] [--gpu] [--mem=<n_gb>]\n", argv[0]);
            return 1;
        }
    }

    if (!run_cpu && !run_gpu) {
        fprintf(stderr, "Error: At least one of --cpu or --gpu must be specified.\n");
        return 1;
    }

    alloc_mem(n_gb);

    pthread_t gpu_thread;
    if (run_gpu) {
        // Initialize CUDA
        int deviceCount;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            return 1;
        }
        if (deviceCount == 0) {
            fprintf(stderr, "No CUDA devices found\n");
            return 1;
        }
        err = hipSetDevice(0);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            return 1;
        }

        // Create GPU torture thread
        if (pthread_create(&gpu_thread, NULL, gpu_torture_thread, NULL) != 0) {
            fprintf(stderr, "Failed to create GPU torture thread\n");
            return 1;
        }
    }

    if (run_cpu) {
        torture_cpu();
    } else if (run_gpu) {
        // If only GPU is running, we need to keep the main thread alive
        while(gpu_torture_running.load()) {
            sleep(1);
        }
    }

    // Clean up
    if (run_gpu) {
        gpu_torture_running.store(false);
        pthread_join(gpu_thread, NULL);
    }

    return 0;
}
