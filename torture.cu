#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

#define BLOCK_SIZE 256
#define NUM_BLOCKS 1024

void *infinite_loop(void *unused) {
  while (1)
    ;
}

char *store_mem = NULL;

void alloc_mem(int n_gb) {
  if (n_gb <= 0)
    return;

  size_t n_b = (size_t)n_gb * 1000 * 1000 * 1000;
  store_mem = (char *)calloc(1, n_b);
  if (!store_mem)
    puts("malloc() failed."), exit(1);

  for (size_t i = 0; i < n_b; i++)
    store_mem[i] = 0;
}

__global__ void gpu_torture_kernel(float *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = 1.0f;
        for (int i = 0; i < 1000000; i++) {
            x = sinf(x) * cosf(x) * sqrtf(x);
        }
        data[idx] = x;
    }
}

void launch_gpu_torture() {
    int n = BLOCK_SIZE * NUM_BLOCKS;
    float *d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(float)));

    dim3 block(BLOCK_SIZE);
    dim3 grid(NUM_BLOCKS);

    while (1) {
        gpu_torture_kernel<<<grid, block>>>(d_data, n);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // This will never be reached, but good practice
    CUDA_CHECK(hipFree(d_data));
}

void load_threads(void) {
    int numThreads = sysconf(_SC_NPROCESSORS_ONLN) - 1; // Get the number of processors.
    pthread_t threads[numThreads];
    int rc;
    for (size_t t = 0; t < numThreads; t++) {
        rc = pthread_create(&threads[t], NULL, infinite_loop, NULL);
        if (rc) {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(1);
        }
    }

    // Launch GPU torture in the main thread
    launch_gpu_torture();
}

int main(int argc, char **argv) {
    int n_gb = -1;
    if (argc < 2)
        n_gb = 0;
    else if (argc == 2)
        n_gb = atoi(argv[1]);
    else
        return puts("Usage: load_all_threads <n_gb>"), 1;

    // Initialize CUDA
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        exit(1);
    }
    CUDA_CHECK(hipSetDevice(0));

    alloc_mem(n_gb);
    load_threads();
    return 0;
}
