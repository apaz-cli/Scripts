#include "hip/hip_runtime.h"
#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <vector>

/*******/
/* GPU */
/*******/

#define BLOCK_SIZE 256
#define HALF_GB (size_t)(512 * 1024 * 1024)

#define CUDA_CHECK(call) \
    do { \
        hipError_t _cu_error = call; \
        if (_cu_error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(_cu_error)); \
            return _cu_error; \
        } \
    } while(0)

__global__ void gpu_torture_kernel(float *data, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = 1.0f;
        for (int i = 0; i < 1000000; i++) {
            x = sinf(x) * cosf(x) * sqrtf(x);
        }
        data[idx] = x;
    }
}

hipError_t launch_gpu_torture() {
    std::vector<float*> gpu_memory_blocks;
    size_t total_allocated = 0;
    
    while (true) {
        float* d_data;
        hipError_t err = hipMalloc(&d_data, HALF_GB);
        if (err != hipSuccess) {
            if (err == hipError_t::cudaErrorOutOfMemory) {
                break;  // We've used all available memory
            }
            return err;
        }
        
        gpu_memory_blocks.push_back(d_data);
        total_allocated += HALF_GB;
    }
    
    printf("Total GPU memory allocated: %.2f GB\n", total_allocated / (1024.0 * 1024.0 * 1024.0));
    
    size_t n = total_allocated / sizeof(float);
    dim3 block(BLOCK_SIZE);
    dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    bool continue_torture = true;
    while (continue_torture) {
        for (float* d_data : gpu_memory_blocks) {
            gpu_torture_kernel<<<grid, block>>>(d_data, n);
            CUDA_CHECK(hipGetLastError());
        }
        CUDA_CHECK(hipDeviceSynchronize());
        puts("GPU torture kernel finished");
        
        // Add a condition to potentially break the loop
        // For example, you could check for a specific file's existence
        // if (file_exists("stop_torture")) {
        //     continue_torture = false;
        // }
    }
    
    // We never reach here, but for completeness:
    for (float* d_data : gpu_memory_blocks) {
        hipFree(d_data);
    }
    
    return hipSuccess;
}

/*******/
/* CPU */
/*******/

void *infinite_loop(void *unused) {
  while (1)
    ;
}

char *store_mem = NULL;

void alloc_mem(int n_gb) {
  if (n_gb <= 0)
    return;

  size_t n_b = (size_t)n_gb * 1000 * 1000 * 1000;
  store_mem = (char *)calloc(1, n_b);
  if (!store_mem)
    puts("malloc() failed."), exit(1);

  for (size_t i = 0; i < n_b; i++)
    store_mem[i] = 0;
}


void torture_cpu(void) {
    int numThreads = sysconf(_SC_NPROCESSORS_ONLN) - 1; // Get the number of processors.
    pthread_t threads[numThreads];
    int rc;
    for (size_t t = 0; t < numThreads; t++) {
        rc = pthread_create(&threads[t], NULL, infinite_loop, NULL);
        if (rc) {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(1);
        }
    }

    // Launch GPU torture in the main thread
    launch_gpu_torture();
}

int main(int argc, char **argv) {
    int n_gb = -1;
    if (argc < 2)
        n_gb = 0;
    else if (argc == 2)
        n_gb = atoi(argv[1]);
    else
        return puts("Usage: load_all_threads <n_gb>"), 1;

    // Initialize CUDA
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        exit(1);
    }
    CUDA_CHECK(hipSetDevice(0));

    alloc_mem(n_gb);
    
    hipError_t result = launch_gpu_torture();
    if (result != hipSuccess) {
        fprintf(stderr, "GPU torture failed: %s\n", hipGetErrorString(result));
        return 1;
    }
    
    torture_cpu();
    return 0;
}
