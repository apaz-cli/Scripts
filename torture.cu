
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>

void *infinite_loop(void *unused) {
  while (1)
    ;
}

char *store_mem = NULL;

void alloc_mem(int n_gb) {
  if (n_gb <= 0)
    return;

  size_t n_b = (size_t)n_gb * 1000 * 1000 * 1000;
  store_mem = (char *)calloc(1, n_b);
  if (!store_mem)
    puts("malloc() failed."), exit(1);

  for (size_t i = 0; i < n_b; i++)
    store_mem[i] = 0;
}

void load_threads(void) {
  int numThreads =
      sysconf(_SC_NPROCESSORS_ONLN) - 1; // Get the number of processors.
  pthread_t threads[numThreads];
  int rc;
  for (size_t t = 0; t < numThreads; t++) {
    rc = pthread_create(&threads[t], NULL, infinite_loop, NULL);
    if (rc) {
      printf("ERROR; return code from pthread_create() is %d\n", rc);
      exit(1);
    }
  }

  infinite_loop(NULL);
}

int main(int argc, char **argv) {

  int n_gb = -1;
  if (argc < 2)
    n_gb = 0;
  else if (argc == 2)
    n_gb = atoi(argv[1]);
  else
    return puts("Usage: load_all_threads <n_gb>"), 1;

  alloc_mem(n_gb);
  load_threads();
  return 0;
}
