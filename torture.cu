#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string.h>
/*******/
/* GPU */
/*******/

#define BLOCK_SIZE 256
#define HALF_GB (size_t)(512 * 1024 * 1024)

#define CUDA_CHECK(call) \
    do { \
        hipError_t _cu_error = call; \
        if (_cu_error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(_cu_error)); \
            return _cu_error; \
        } \
    } while(0)

__global__ void gpu_torture_kernel(float **data_blocks, size_t n_blocks, size_t block_size) {
    size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total_elements = n_blocks * block_size;
    
    if (thread_id < total_elements) {
        size_t block_idx = thread_id / block_size;
        size_t element_idx = thread_id % block_size;
        
        float x = 1.0f;
        float y = 2.0f;
        float z = 3.0f;
        
        for (int i = 0; i < 1000000; i++) {
            x = sinf(x) * cosf(y) * tanf(z);
            y = expf(x) * logf(fabsf(y)) * sqrtf(fabsf(z));
            z = powf(x, y) * fmodf(z, 3.14159f);
            
            // Access and modify data from different blocks
            for (size_t j = 0; j < n_blocks; j++) {
                size_t idx = (block_idx + j) % n_blocks;
                float* block = data_blocks[idx];
                float val = block[(element_idx + j) % block_size];
                x += val;
                y *= val;
                z -= val;
            }
        }
        
        // Write results back to all blocks
        for (size_t j = 0; j < n_blocks; j++) {
            size_t idx = (block_idx + j) % n_blocks;
            float* block = data_blocks[idx];
            block[(element_idx + j) % block_size] = x + y + z;
        }
    }
}

void* launch_gpu_torture(void* arg) {
  (void)arg;
    std::vector<float*> gpu_memory_blocks;
    size_t total_allocated = 0;
    
    while (true) {
        float* d_data;
        hipError_t err = hipMalloc(&d_data, HALF_GB);
        if (err != hipSuccess) {
            if (err == hipErrorOutOfMemory) {
                hipGetLastError(); // Clear the error
                break;  // We've used all available memory
            }
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            return;
        }
        
        gpu_memory_blocks.push_back(d_data);
        total_allocated += HALF_GB;
    }
    
    printf("Total GPU memory allocated: %.2f GB\n", total_allocated / (1024.0 * 1024.0 * 1024.0));
    
    size_t n_blocks = gpu_memory_blocks.size();
    size_t block_size = HALF_GB / sizeof(float);
    size_t total_elements = n_blocks * block_size;
    
    // Allocate and copy device pointers to GPU
    float **d_data_blocks;
    if (hipMalloc(&d_data_blocks, n_blocks * sizeof(float*)) != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for data blocks\n");
        return;
    }
    if (hipMemcpy(d_data_blocks, gpu_memory_blocks.data(), n_blocks * sizeof(float*), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy data blocks to device\n");
        return;
    }
    
    dim3 block(BLOCK_SIZE);
    dim3 grid((total_elements + BLOCK_SIZE - 1) / BLOCK_SIZE);

    gpu_torture_kernel<<<grid, block>>>(d_data_blocks, n_blocks, block_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    } else {
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        } else {
            puts("GPU torture kernel finished");
        }
    }

    // Free allocated memory
    for (auto block : gpu_memory_blocks) {
        hipFree(block);
    }
    hipFree(d_data_blocks);
}

/*******/
/* CPU */
/*******/

void *infinite_loop(void *unused) {
  while (1)
    ;
}

char *store_mem = NULL;

void alloc_mem(size_t n_bytes) {
  if (n_bytes == 0)
    return;

  size_t n_b = n_bytes;
  store_mem = (char *)calloc(1, n_b);
  if (!store_mem)
    puts("malloc() failed."), exit(1);

  for (size_t i = 0; i < n_b; i++)
    store_mem[i] = 0;
}


void torture_cpu(void) {
    // Create one minus the number of CPUs threads. The last one is this thread.
    int numThreads = sysconf(_SC_NPROCESSORS_ONLN) - 1; 
    pthread_t threads[numThreads];
    int rc;
    for (size_t t = 0; t < numThreads; t++) {
        rc = pthread_create(&threads[t], NULL, infinite_loop, NULL);
        if (rc) {
            printf("ERROR; return code from pthread_create() is %d\n", rc);
            exit(1);
        }
    }

    infinite_loop(NULL);
}

int main(int argc, char **argv) {
    size_t mem_bytes = 0;
    bool run_cpu = false;
    bool run_gpu = false;

    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--cpu") == 0) {
            run_cpu = true;
        } else if (strcmp(argv[i], "--gpu") == 0) {
            run_gpu = true;
        } else if (strncmp(argv[i], "--mem=", 6) == 0) {
            char* mem_str = argv[i] + 6;
            char* endptr;
            double mem_value = strtod(mem_str, &endptr);
            if (mem_value <= 0 || mem_str == endptr) {
                fprintf(stderr, "Invalid memory value: %s\n", mem_str);
                return 1;
            }
            if (*endptr == '\0' || strcasecmp(endptr, "B") == 0) {
                mem_bytes = (size_t)mem_value;
            } else if (strcasecmp(endptr, "K") == 0 || strcasecmp(endptr, "KB") == 0) {
                mem_bytes = (size_t)(mem_value * 1024);
            } else if (strcasecmp(endptr, "M") == 0 || strcasecmp(endptr, "MB") == 0) {
                mem_bytes = (size_t)(mem_value * 1024 * 1024);
            } else if (strcasecmp(endptr, "G") == 0 || strcasecmp(endptr, "GB") == 0) {
                mem_bytes = (size_t)(mem_value * 1024 * 1024 * 1024);
            } else {
                fprintf(stderr, "Invalid memory unit: %s\n", endptr);
                return 1;
            }
        } else {
            fprintf(stderr, "Usage: %s [--cpu] [--gpu] [--mem=<size>[B|K|M|G]]\n", argv[0]);
            return 1;
        }
    }

    if (!run_cpu && !run_gpu) {
        fprintf(stderr, "Error: At least one of --cpu or --gpu must be specified.\n");
        return 1;
    }

    alloc_mem(mem_bytes);

    pthread_t gpu_thread;
    if (run_gpu) {
        // Initialize CUDA
        int deviceCount;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            return 1;
        }
        if (deviceCount == 0) {
            fprintf(stderr, "No CUDA devices found\n");
            return 1;
        }
        err = hipSetDevice(0);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
            return 1;
        }

        // Create GPU torture thread
        if (pthread_create(&gpu_thread, NULL, launch_gpu_torture, NULL) != 0) {
            fprintf(stderr, "Failed to create GPU torture thread\n");
            return 1;
        }
    }

    if (run_cpu) {
        torture_cpu();
    }

    return 0;
}
